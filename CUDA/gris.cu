
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>

#define CHANNELS 3

using namespace std;

//leer imagen
unsigned char* readBMP(char* filename, int &my_width, int &my_height)
{
    int i;
    FILE* f = fopen(filename, "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];

    int size = 3 * width * height;
    unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
    fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
    fclose(f);
    my_width = width;
	my_height = height;

    return data;
}

//guardar imagen
void writeBMP(unsigned char* img, int w, int h)
{
    FILE *f;
    int filesize = 54 + 3*w*h;  //w is your image width, h is image height, both int
    unsigned char bmpfileheader[14] = {'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0};
    unsigned char bmpinfoheader[40] = {40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0};
    unsigned char bmppad[3] = {0,0,0};
    bmpfileheader[ 2] = (unsigned char)(filesize    );
    bmpfileheader[ 3] = (unsigned char)(filesize>> 8);
    bmpfileheader[ 4] = (unsigned char)(filesize>>16);
    bmpfileheader[ 5] = (unsigned char)(filesize>>24);
    bmpinfoheader[ 4] = (unsigned char)(       w    );
    bmpinfoheader[ 5] = (unsigned char)(       w>> 8);
    bmpinfoheader[ 6] = (unsigned char)(       w>>16);
    bmpinfoheader[ 7] = (unsigned char)(       w>>24);
    bmpinfoheader[ 8] = (unsigned char)(       h    );
    bmpinfoheader[ 9] = (unsigned char)(       h>> 8);
    bmpinfoheader[10] = (unsigned char)(       h>>16);
    bmpinfoheader[11] = (unsigned char)(       h>>24);
    f = fopen("img.bmp","wb");
    fwrite(bmpfileheader,1,14,f);
    fwrite(bmpinfoheader,1,40,f);
    //for(int i=0; i<h; i++)
    for(int i=h-1; i>=0; i--)
    {
        fwrite(img+(w*(h-i-1)*3),3,w,f);
        fwrite(bmppad,1,(4-(w*3)%4)%4,f);
    }
    free(img);
    fclose(f);
}


// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__
void colorToGreyscaleConversion(unsigned char * Pout, unsigned char * Pin, int width, int height)
{
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;
    if (Col < width && Row < height)
    {
		// get 1D coordinate for the grayscale image
		int greyOffset = Row*width + Col;
		// one can think of the RGB image having
		// CHANNEL times columns than the grayscale image
		int rgbOffset = greyOffset*CHANNELS;
		unsigned char r = Pin[rgbOffset ]; // red value for pixel
		unsigned char g = Pin[rgbOffset + 2]; // green value for pixel
		unsigned char b = Pin[rgbOffset + 3]; // blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}

int main()
{
    unsigned char* h_imagen_in;
	unsigned char* h_imagen_out;
	unsigned char* d_imagen_in;
	unsigned char* d_imagen_out;
	int width = 0;
	int height = 0;
	string archivo;
	//cout<<"Ingrese nombre de la imagen: ";
	//cin>>archivo;
	
	h_imagen_in = readBMP("Tulips.bmp", width, height);
	
	int size = width * height * sizeof(unsigned char);
	
	h_imagen_out = (unsigned char*)malloc(size * sizeof(unsigned char));
	
	hipMalloc((void **) &d_imagen_in, size*3);
    hipMemcpy(d_imagen_in, h_imagen_in, size*3, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_imagen_out, size);
    hipMemcpy(d_imagen_out, h_imagen_out, size, hipMemcpyHostToDevice);
	
	dim3 dimGrid(ceil(width/16.0), ceil(height/16.0), 1);
	dim3 dimBlock(16, 16, 1);
	colorToGreyscaleConversion<<<dimGrid,dimBlock>>>(d_imagen_out,d_imagen_in,width,height);
	hipMemcpy(h_imagen_out, d_imagen_out, size, hipMemcpyDeviceToHost);
	
	writeBMP(h_imagen_out, width, height);
    return 0;
}