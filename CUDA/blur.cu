
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>

#define BLUR_SIZE 1

using namespace std;

//leer imagen
unsigned char* readBMP(char* filename, int &my_width, int &my_height)
{
    int i;
    FILE* f = fopen(filename, "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];

    int size = 3 * width * height;
    unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
    fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
    fclose(f);
    my_width = width;
	my_height = height;

    return data;
}

//guardar imagen
void writeBMP(unsigned char* img, int w, int h)
{
    FILE *f;
    int filesize = 54 + 3*w*h;  //w is your image width, h is image height, both int
    unsigned char bmpfileheader[14] = {'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0};
    unsigned char bmpinfoheader[40] = {40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0};
    unsigned char bmppad[3] = {0,0,0};
    bmpfileheader[ 2] = (unsigned char)(filesize    );
    bmpfileheader[ 3] = (unsigned char)(filesize>> 8);
    bmpfileheader[ 4] = (unsigned char)(filesize>>16);
    bmpfileheader[ 5] = (unsigned char)(filesize>>24);
    bmpinfoheader[ 4] = (unsigned char)(       w    );
    bmpinfoheader[ 5] = (unsigned char)(       w>> 8);
    bmpinfoheader[ 6] = (unsigned char)(       w>>16);
    bmpinfoheader[ 7] = (unsigned char)(       w>>24);
    bmpinfoheader[ 8] = (unsigned char)(       h    );
    bmpinfoheader[ 9] = (unsigned char)(       h>> 8);
    bmpinfoheader[10] = (unsigned char)(       h>>16);
    bmpinfoheader[11] = (unsigned char)(       h>>24);
    f = fopen("img.bmp","wb");
    fwrite(bmpfileheader,1,14,f);
    fwrite(bmpinfoheader,1,40,f);
    //for(int i=0; i<h; i++)
    for(int i=h-1; i>=0; i--)
    {
        fwrite(img+(w*(h-i-1)*3),3,w,f);
        fwrite(bmppad,1,(4-(w*3)%4)%4,f);
    }
    free(img);
    fclose(f);
}


__global__
void blurKernel(unsigned char *in, unsigned char *out, int w, int h)
{
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(Col<w && Row<h)
	{
		int pixVal = 0;
        int pixels = 0;
        
        for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
        {
            for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
            {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;
                
                if(curRow>-1 && curRow<h && curCol>-1 && curCol<w)
                {
                    pixVal += in[curRow * w +curCol];
                    pixels++;
                }
            }
            out[Row * w + Col] = (unsigned char)(pixVal/pixels);
        }
	}
	
	
}

int main()
{
    unsigned char* imagen_in;
	unsigned char* imagen_out;
	int width = 0;
	int height = 0;
	string archivo;
	//cout<<"Ingrese nombre de la imagen: ";
	//cin>>archivo;
	
	imagen_in = readBMP("Tulips.bmp", width, height);
	
	dim3 dimGrid(ceil(width/16.0), ceil(height/16.0), 1);
	dim3 dimBlock(16, 16, 1);
	blurKernel<<<dimGrid,dimBlock>>>(imagen_in,imagen_out,width,height);
	
	writeBMP(imagen_out, width, height);

    return 0;
}