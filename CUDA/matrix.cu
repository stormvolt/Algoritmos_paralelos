
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>


#define ITER 100
#define TILE_WIDTH 4

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width)
{
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;
	// Loop over the d_M and d_N tiles required to compute d_P element
	for (int ph = 0; ph < Width/TILE_WIDTH; ++ph)
	{
		// Collaborative loading of d_M and d_N tiles into shared memory
		Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
		Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
		{
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	d_P[Row*Width + Col] = Pvalue;
}


int main()
{
    float *h_a, *h_b, *h_c;
    h_a = (float *)malloc(ITER * sizeof(float));
    h_b = (float *)malloc(ITER * sizeof(float));
    h_c = (float *)malloc(ITER * sizeof(float));

    for (int i = 0; i < ITER; ++i)
    {
        h_a[i] = i;
        h_b[i] = i;
        h_c[i] = i;
    }

    MatrixMulKernel<<<ceil(ITER/256.0), 256>>>(h_a, h_b, h_c, ITER);
	
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
